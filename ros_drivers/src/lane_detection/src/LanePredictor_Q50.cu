#include "hip/hip_runtime.h"
#include "LanePredictor_Q50.h"

#define BATCH_SIZE 1
#define IMG_DIMZ 3
#define IMG_DIMX 635
#define IMG_DIMY 363
#define IMG_OFFSETX 5
#define IMG_OFFSETY 117

#define MEAN_AXIS1 2
#define MEAN_AXIS2 3
#define MEAN_SIZE1 13
#define MEAN_SIZE2 13
#define MEAN_SIGMA ((MEAN_SIZE1-1)/4.0)
#define MEAN_STEP1 1
#define MEAN_STEP2 1
#define MEAN_OUTPUTZ IMG_DIMZ
#define MEAN_OUTPUTX ((IMG_DIMX-MEAN_SIZE1)/MEAN_STEP1+1)
#define MEAN_OUTPUTY ((IMG_DIMY-MEAN_SIZE2)/MEAN_STEP2+1)
#define MEAN_POS DDim(0,(IMG_DIMZ-MEAN_OUTPUTZ)/2,(IMG_DIMX-MEAN_OUTPUTX)/2,(IMG_DIMY-MEAN_OUTPUTY)/2)
#define DIVIDE_OUTPUTZ IMG_DIMZ
#define DIVIDE_OUTPUTX ((MEAN_OUTPUTX-MEAN_SIZE1)/MEAN_STEP1+1)
#define DIVIDE_OUTPUTY ((MEAN_OUTPUTY-MEAN_SIZE2)/MEAN_STEP2+1)
#define DIVIDE_POS DDim(0,(MEAN_OUTPUTZ-DIVIDE_OUTPUTZ)/2,(MEAN_OUTPUTX-DIVIDE_OUTPUTX)/2,(MEAN_OUTPUTY-DIVIDE_OUTPUTY)/2)
#define DIVIDE_EPS 500.0  

#define MAPS_1 16
#define FILT1_DIMZ 3
#define FILT1_DIMX 13
#define FILT1_DIMY 13
#define FILT1_STEPZ 3
#define FILT1_STEPX 2
#define FILT1_STEPY 2
#define FILT1_OUTPUTZ (((DIVIDE_OUTPUTZ-FILT1_DIMZ)/FILT1_STEPZ+1)*MAPS_1)
#define FILT1_OUTPUTX ((DIVIDE_OUTPUTX-FILT1_DIMX)/FILT1_STEPX+1)
#define FILT1_OUTPUTY ((DIVIDE_OUTPUTY-FILT1_DIMY)/FILT1_STEPY+1)
#define THRESHOLD_NL "fmaxf(.1f*a,a)"
#define SUBSAMP1_DIMZ MAPS_1
#define SUBSAMP1_DIMX 3
#define SUBSAMP1_DIMY 3
#define SUBSAMP1_STEPZ MAPS_1
#define SUBSAMP1_STEPX 3
#define SUBSAMP1_STEPY 3
#define SUBSAMP1_OUTPUTZ (((FILT1_OUTPUTZ-SUBSAMP1_DIMZ)/SUBSAMP1_STEPZ+1)*MAPS_1)
#define SUBSAMP1_OUTPUTX ((FILT1_OUTPUTX-SUBSAMP1_DIMX)/SUBSAMP1_STEPX+1)
#define SUBSAMP1_OUTPUTY ((FILT1_OUTPUTY-SUBSAMP1_DIMY)/SUBSAMP1_STEPY+1)

#define MAPS_2 16
#define FILT2_DIMZ MAPS_1
#define FILT2_DIMX 7
#define FILT2_DIMY 7
#define FILT2_STEPZ MAPS_2
#define FILT2_STEPX 1
#define FILT2_STEPY 1
#define FILT2_OUTPUTZ (((SUBSAMP1_OUTPUTZ-FILT2_DIMZ)/FILT2_STEPZ+1)*MAPS_2)
#define FILT2_OUTPUTX ((SUBSAMP1_OUTPUTX-FILT2_DIMX)/FILT2_STEPX+1)
#define FILT2_OUTPUTY ((SUBSAMP1_OUTPUTY-FILT2_DIMY)/FILT2_STEPY+1)
#define SUBSAMP2_DIMZ MAPS_2
#define SUBSAMP2_DIMX 2
#define SUBSAMP2_DIMY 2
#define SUBSAMP2_STEPZ MAPS_2
#define SUBSAMP2_STEPX 2
#define SUBSAMP2_STEPY 2
#define SUBSAMP2_OUTPUTZ (((FILT2_OUTPUTZ-SUBSAMP2_DIMZ)/SUBSAMP2_STEPZ+1)*MAPS_2)
#define SUBSAMP2_OUTPUTX ((FILT2_OUTPUTX-SUBSAMP2_DIMX)/SUBSAMP2_STEPX+1)
#define SUBSAMP2_OUTPUTY ((FILT2_OUTPUTY-SUBSAMP2_DIMY)/SUBSAMP2_STEPY+1)

#define MAPS_3 32
#define FILT3_DIMZ MAPS_2
#define FILT3_DIMX 5
#define FILT3_DIMY 5
#define FILT3_STEPZ MAPS_3
#define FILT3_STEPX 1
#define FILT3_STEPY 1
#define FILT3_OUTPUTZ (((SUBSAMP2_OUTPUTZ-FILT3_DIMZ)/FILT3_STEPZ+1)*MAPS_3)
#define FILT3_OUTPUTX ((SUBSAMP2_OUTPUTX-FILT3_DIMX)/FILT3_STEPX+1)
#define FILT3_OUTPUTY ((SUBSAMP2_OUTPUTY-FILT3_DIMY)/FILT3_STEPY+1)

#define BOTTLENECK 960

#define NUM_CLASSES 1
#define NUM_CLASSIFIERS 40

#define MEAN_LCN_SPLITS 16
#define DIVIDE_LCN_SPLITS 16
#define W1_SPLITS 16
#define W2_SPLITS 16
#define W3_SPLITS 16

LanePredictor_Q50::LanePredictor_Q50(int* argc, char* argv[],int stream_num){
	stream = stream_num;
	model = argv[1];

	data_buf = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,IMG_DIMZ,IMG_DIMX,IMG_DIMY), stream);
	mean_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,MEAN_OUTPUTZ,MEAN_OUTPUTX,MEAN_OUTPUTY),stream);
    mean_lcn_buf = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,MEAN_OUTPUTZ,MEAN_OUTPUTX,IMG_DIMY),stream);
    mean_lcn_sqr = gpuArrayAllocRM(DataType::FLOAT, mean_lcn->dim(),stream);
	divide_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,DIVIDE_OUTPUTZ,DIVIDE_OUTPUTX,DIVIDE_OUTPUTY),stream);
    divide_lcn_buf = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,DIVIDE_OUTPUTZ,DIVIDE_OUTPUTX,MEAN_OUTPUTY),stream);
	filt_LCN = gpuArrayAllocRM(DataType::FLOAT,DDim(IMG_DIMZ,1,1,IMG_DIMZ,MEAN_SIZE1,MEAN_SIZE2),stream);
    filt_LCN_X = gpuArrayAllocRM(DataType::FLOAT,DDim(MEAN_SIZE1),stream);
	init_filt_LCN(filt_LCN,MEAN_SIGMA,stream);
    init_filt_LCN_sep(filt_LCN_X,MEAN_SIGMA,stream);
    gpuTimesScalar(filt_LCN_X,1.0/IMG_DIMZ,filt_LCN_X,stream);

	W_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_1,1,1,FILT1_DIMZ,FILT1_DIMX,FILT1_DIMY), stream);
	Ptr<DistArrayHandle> W_1_load = loadDistArray(model+"_W_1",DDim(1,1,1,1,1,1),stream);
	copy(W_1_load->localHandle(),W_1,stream);
	filt_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT1_OUTPUTZ,FILT1_OUTPUTX,FILT1_OUTPUTY), stream);
	nonlin_1 = gpuArrayAllocRM(DataType::FLOAT, filt_1->dim(),stream);
	pool_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP1_OUTPUTZ,SUBSAMP1_OUTPUTX,SUBSAMP1_OUTPUTY),stream);

	W_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_2,1,1,FILT2_DIMZ,FILT2_DIMX,FILT2_DIMY),stream);
	Ptr<DistArrayHandle> W_2_load = loadDistArray(model+"_W_2",DDim(1,1,1,1,1,1),stream);
	copy(W_2_load->localHandle(),W_2,stream);
	filt_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT2_OUTPUTZ,FILT2_OUTPUTX,FILT2_OUTPUTY),stream);
	nonlin_2 = gpuArrayAllocRM(DataType::FLOAT, filt_2->dim(),stream);
	pool_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP2_OUTPUTZ,SUBSAMP2_OUTPUTX,SUBSAMP2_OUTPUTY),stream);

	W_3 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_3,1,1,FILT3_DIMZ,FILT3_DIMX,FILT3_DIMY),stream);
	Ptr<DistArrayHandle> W_3_load = loadDistArray(model+"_W_3",DDim(1,1,1,1,1,1),stream);
	copy(W_3_load->localHandle(),W_3,stream);
	feat = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT3_OUTPUTZ,FILT3_OUTPUTX,FILT3_OUTPUTY),stream);

	W_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BOTTLENECK,feat->dim(1),feat->dim(2),feat->dim(3)),stream);
	Ptr<DistArrayHandle> W_bn_load = loadDistArray(model+"_W_bn",DDim(1,1,1,1),stream);
	copy(W_bn_load->localHandle(),W_bn,stream);
	B_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(1,BOTTLENECK),stream);
	Ptr<DistArrayHandle> B_bn_load = loadDistArray(model+"_B_bn",DDim(1,1,1,1),stream);
	copy(B_bn_load->localHandle(),B_bn,stream);
	feat_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,BOTTLENECK),stream);
    feat_bn_nonlin = gpuArrayAllocRM(DataType::FLOAT,feat_bn->dim(),stream);

	W_final = gpuArrayAllocRM(DataType::FLOAT,DDim(NUM_CLASSES*NUM_CLASSIFIERS,BOTTLENECK),stream);
	Ptr<DistArrayHandle> W_final_load = loadDistArray(model+"_W_final",DDim(1,1,1,1),stream);
	copy(W_final_load->localHandle(),W_final,stream);
	B_final = gpuArrayAllocRM(DataType::FLOAT,DDim(1,W_final->dim(0)),stream);
	Ptr<DistArrayHandle> B_final_load = loadDistArray(model+"_B_final",DDim(1,1,1,1),stream);
	copy(B_final_load->localHandle(),B_final,stream);
	mult_linear = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,W_final->dim(0)),stream);

	allocScratchArray(data_buf,mean_lcn,filt_LCN,MEAN_STEP2,data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
	allocScratchArray(mean_lcn_sqr,divide_lcn,filt_LCN,MEAN_STEP2,mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
	allocScratchArray(divide_lcn,filt_1,W_1,FILT1_STEPY,divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	allocScratchArray(pool_1,filt_2,W_2,FILT2_STEPY,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	allocScratchArray(pool_2,feat,W_3,FILT3_STEPY,pool_2_scratch,feat_scratch,stream,W3_SPLITS);
    
    host_output = hostArrayAllocRM(DataType::FLOAT,DDim(1,NUM_CLASSIFIERS),stream);

	synchronizeStream(stream);
}
LanePredictor_Q50::~LanePredictor_Q50(){
}

Ptr<ArrayViewHandle> LanePredictor_Q50::processImage(const Ptr<ArrayViewHandle>& img){
    for(int i=IMG_OFFSETY;i<(IMG_OFFSETY+IMG_DIMY);i++){
        copy(img->view(DDim(0,0,IMG_OFFSETX,i),DDim(BATCH_SIZE,IMG_DIMZ,IMG_DIMX,1)),
            data_buf->view(DDim(0,0,0,i-IMG_OFFSETY),DDim(BATCH_SIZE,IMG_DIMZ,IMG_DIMX,1)),stream);
    }
	//gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,data_buf,mean_lcn,
	//	data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
    gpuSeparable2DFilterTimes(filt_LCN_X,filt_LCN_X,true,MEAN_STEP1,MEAN_STEP2,data_buf,mean_lcn_buf,mean_lcn,0);
	gpuMinus(data_buf->view(MEAN_POS,mean_lcn->dim()),mean_lcn,mean_lcn,stream);

	gpuSquare(mean_lcn,mean_lcn_sqr,stream);
	//gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,mean_lcn_sqr,divide_lcn,
	//	mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
    gpuSeparable2DFilterTimes(filt_LCN_X,filt_LCN_X,true,MEAN_STEP1,MEAN_STEP2,mean_lcn_sqr,divide_lcn_buf,divide_lcn,0);
	gpuPlusScalar(divide_lcn,DIVIDE_EPS,divide_lcn,stream);
	gpuSqrt(divide_lcn,divide_lcn,stream);
	gpuDivide(mean_lcn->view(DIVIDE_POS,divide_lcn->dim()),divide_lcn,divide_lcn,stream);

	gpuFilterTimesLarge(W_1,false,FILT1_STEPZ,FILT1_STEPX,FILT1_STEPY,divide_lcn,filt_1,
		divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	gpuBinaryOp(filt_1,filt_1,nonlin_1,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_1,2,3,SUBSAMP1_DIMX,SUBSAMP1_DIMY,SUBSAMP1_STEPX,SUBSAMP1_STEPY,pool_1,stream);
	gpuTimesScalar(pool_1,1.0/(SUBSAMP1_DIMX*SUBSAMP1_DIMY),pool_1,stream);

	gpuFilterTimesLarge(W_2,false,FILT2_STEPZ,FILT2_STEPX,FILT2_STEPY,pool_1,filt_2,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	gpuBinaryOp(filt_2,filt_2,nonlin_2,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_2,2,3,SUBSAMP2_DIMX,SUBSAMP2_DIMY,SUBSAMP2_STEPX,SUBSAMP2_STEPY,pool_2,stream);
	gpuTimesScalar(pool_2,1.0/(SUBSAMP2_DIMX*SUBSAMP2_DIMY),pool_2,stream);

	gpuFilterTimesLarge(W_3,false,FILT3_STEPZ,FILT3_STEPX,FILT3_STEPY,pool_2,feat,pool_2_scratch,feat_scratch,stream,W3_SPLITS);

	gpuGEMM('n','t',1.0f,reshape(feat,DDim(BATCH_SIZE,feat->dim(1)*feat->dim(2)*feat->dim(3)),stream),
		reshape(W_bn,DDim(BOTTLENECK,W_bn->dim(1)*W_bn->dim(2)*W_bn->dim(3)),stream),0.0f,feat_bn,stream);
	gpuPlus(feat_bn,B_bn,feat_bn,stream);

    gpuBinaryOp(feat_bn,feat_bn,feat_bn_nonlin,THRESHOLD_NL,stream);

	gpuGEMM('n','t',1.0f,feat_bn_nonlin,W_final,0.0f,mult_linear,stream);
	gpuPlus(mult_linear,B_final,mult_linear,stream);

    copy(mult_linear,host_output,stream);

	return host_output;
}
