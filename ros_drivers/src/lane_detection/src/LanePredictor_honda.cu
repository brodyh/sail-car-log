#include "hip/hip_runtime.h"
#include "LanePredictor_honda.h"

#define BATCH_SIZE 1
#define IMG_DIMZ 3
#define IMG_DIMX 640
#define IMG_DIMY 480

#define MEAN_AXIS1 2
#define MEAN_AXIS2 3
#define MEAN_SIZE1 9
#define MEAN_SIZE2 9
#define MEAN_SIGMA ((MEAN_SIZE1-1)/4.0)
#define MEAN_STEP1 1
#define MEAN_STEP2 1
#define MEAN_OUTPUTZ IMG_DIMZ
#define MEAN_OUTPUTX ((IMG_DIMX-MEAN_SIZE1)/MEAN_STEP1+1)
#define MEAN_OUTPUTY ((IMG_DIMY-MEAN_SIZE2)/MEAN_STEP2+1)
#define MEAN_POS DDim(0,(IMG_DIMZ-MEAN_OUTPUTZ)/2,(IMG_DIMX-MEAN_OUTPUTX)/2,(IMG_DIMY-MEAN_OUTPUTY)/2)
#define DIVIDE_OUTPUTZ IMG_DIMZ
#define DIVIDE_OUTPUTX ((MEAN_OUTPUTX-MEAN_SIZE1)/MEAN_STEP1+1)
#define DIVIDE_OUTPUTY ((MEAN_OUTPUTY-MEAN_SIZE2)/MEAN_STEP2+1)
#define DIVIDE_POS DDim(0,(MEAN_OUTPUTZ-DIVIDE_OUTPUTZ)/2,(MEAN_OUTPUTX-DIVIDE_OUTPUTX)/2,(MEAN_OUTPUTY-DIVIDE_OUTPUTY)/2)
#define DIVIDE_EPS 500.0    

#define MAPS_1 16
#define FILT1_DIMZ 3
#define FILT1_DIMX 10
#define FILT1_DIMY 10
#define FILT1_STEPZ 3
#define FILT1_STEPX 2
#define FILT1_STEPY 2
#define FILT1_OUTPUTZ (((DIVIDE_OUTPUTZ-FILT1_DIMZ)/FILT1_STEPZ+1)*MAPS_1)
#define FILT1_OUTPUTX ((DIVIDE_OUTPUTX-FILT1_DIMX)/FILT1_STEPX+1)
#define FILT1_OUTPUTY ((DIVIDE_OUTPUTY-FILT1_DIMY)/FILT1_STEPY+1)
#define THRESHOLD_NL "fmaxf(.1f*a,a)"
#define SUBSAMP1_DIMZ MAPS_1
#define SUBSAMP1_DIMX 2
#define SUBSAMP1_DIMY 2
#define SUBSAMP1_STEPZ MAPS_1
#define SUBSAMP1_STEPX 2
#define SUBSAMP1_STEPY 2
#define SUBSAMP1_OUTPUTZ (((FILT1_OUTPUTZ-SUBSAMP1_DIMZ)/SUBSAMP1_STEPZ+1)*MAPS_1)
#define SUBSAMP1_OUTPUTX ((FILT1_OUTPUTX-SUBSAMP1_DIMX)/SUBSAMP1_STEPX+1)
#define SUBSAMP1_OUTPUTY ((FILT1_OUTPUTY-SUBSAMP1_DIMY)/SUBSAMP1_STEPY+1)

#define MAPS_2 16
#define FILT2_DIMZ MAPS_1
#define FILT2_DIMX 9
#define FILT2_DIMY 9
#define FILT2_STEPZ MAPS_2
#define FILT2_STEPX 1
#define FILT2_STEPY 1
#define FILT2_OUTPUTZ (((SUBSAMP1_OUTPUTZ-FILT2_DIMZ)/FILT2_STEPZ+1)*MAPS_2)
#define FILT2_OUTPUTX ((SUBSAMP1_OUTPUTX-FILT2_DIMX)/FILT2_STEPX+1)
#define FILT2_OUTPUTY ((SUBSAMP1_OUTPUTY-FILT2_DIMY)/FILT2_STEPY+1)
#define SUBSAMP2_DIMZ MAPS_2
#define SUBSAMP2_DIMX 2
#define SUBSAMP2_DIMY 2
#define SUBSAMP2_STEPZ MAPS_2
#define SUBSAMP2_STEPX 2
#define SUBSAMP2_STEPY 2
#define SUBSAMP2_OUTPUTZ (((FILT2_OUTPUTZ-SUBSAMP2_DIMZ)/SUBSAMP2_STEPZ+1)*MAPS_2)
#define SUBSAMP2_OUTPUTX ((FILT2_OUTPUTX-SUBSAMP2_DIMX)/SUBSAMP2_STEPX+1)
#define SUBSAMP2_OUTPUTY ((FILT2_OUTPUTY-SUBSAMP2_DIMY)/SUBSAMP2_STEPY+1)

#define MAPS_3 32
#define FILT3_DIMZ MAPS_2
#define FILT3_DIMX 8
#define FILT3_DIMY 8
#define FILT3_STEPZ MAPS_3
#define FILT3_STEPX 1
#define FILT3_STEPY 1
#define FILT3_OUTPUTZ (((SUBSAMP2_OUTPUTZ-FILT3_DIMZ)/FILT3_STEPZ+1)*MAPS_3)
#define FILT3_OUTPUTX ((SUBSAMP2_OUTPUTX-FILT3_DIMX)/FILT3_STEPX+1)
#define FILT3_OUTPUTY ((SUBSAMP2_OUTPUTY-FILT3_DIMY)/FILT3_STEPY+1)

#define BOTTLENECK 480

#define NUM_CLASSES 80
#define NUM_CLASSIFIERS 48
#define MIN_ACTIVATION 1.0e-8
#define MAX_ACTIVATION 1.0-1.0e-8

#define SUBPIXEL_WINDOW 7

#define MEAN_LCN_SPLITS 16
#define DIVIDE_LCN_SPLITS 16
#define W1_SPLITS 16
#define W2_SPLITS 16
#define W3_SPLITS 16



LanePredictor_honda::LanePredictor_honda(int* argc, char* argv[],int stream_num){
	stream = stream_num;
	model = argv[1];
	fastCppInit(argc,&argv,stream);

	data_buf = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,IMG_DIMZ,IMG_DIMX,IMG_DIMY), stream);
	mean_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,MEAN_OUTPUTZ,MEAN_OUTPUTX,MEAN_OUTPUTY),stream);
	mean_lcn_sqr = gpuArrayAllocRM(DataType::FLOAT, mean_lcn->dim(),stream);
	divide_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,DIVIDE_OUTPUTZ,DIVIDE_OUTPUTX,DIVIDE_OUTPUTY),stream);
	filt_LCN = gpuArrayAllocRM(DataType::FLOAT,DDim(IMG_DIMZ,1,1,IMG_DIMZ,MEAN_SIZE1,MEAN_SIZE2),stream);
	init_filt_LCN(filt_LCN,MEAN_SIGMA,stream);

	W_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_1,1,1,FILT1_DIMZ,FILT1_DIMX,FILT1_DIMY), stream);
	Ptr<DistArrayHandle> W_1_load = loadDistArray(model+"_W_1",DDim(1,1,1,1,1,1),stream);
	copy(W_1_load->localHandle(),W_1,stream);
	filt_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT1_OUTPUTZ,FILT1_OUTPUTX,FILT1_OUTPUTY), stream);
	nonlin_1 = gpuArrayAllocRM(DataType::FLOAT, filt_1->dim(),stream);
	pool_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP1_OUTPUTZ,SUBSAMP1_OUTPUTX,SUBSAMP1_OUTPUTY),stream);

	W_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_2,1,1,FILT2_DIMZ,FILT2_DIMX,FILT2_DIMY),stream);
	Ptr<DistArrayHandle> W_2_load = loadDistArray(model+"_W_2",DDim(1,1,1,1,1,1),stream);
	copy(W_2_load->localHandle(),W_2,stream);
	filt_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT2_OUTPUTZ,FILT2_OUTPUTX,FILT2_OUTPUTY),stream);
	nonlin_2 = gpuArrayAllocRM(DataType::FLOAT, filt_2->dim(),stream);
	pool_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP2_OUTPUTZ,SUBSAMP2_OUTPUTX,SUBSAMP2_OUTPUTY),stream);

	W_3 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_3,1,1,FILT3_DIMZ,FILT3_DIMX,FILT3_DIMY),stream);
	Ptr<DistArrayHandle> W_3_load = loadDistArray(model+"_W_3",DDim(1,1,1,1,1,1),stream);
	copy(W_3_load->localHandle(),W_3,stream);
	feat = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT3_OUTPUTZ,FILT3_OUTPUTX,FILT3_OUTPUTY),stream);

	W_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BOTTLENECK,feat->dim(1),feat->dim(2),feat->dim(3)),stream);
	Ptr<DistArrayHandle> W_bn_load = loadDistArray(model+"_W_bn",DDim(1,1,1,1),stream);
	copy(W_bn_load->localHandle(),W_bn,stream);
	B_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(1,BOTTLENECK),stream);
	Ptr<DistArrayHandle> B_bn_load = loadDistArray(model+"_B_bn",DDim(1,1,1,1),stream);
	copy(B_bn_load->localHandle(),B_bn,stream);
	feat_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,BOTTLENECK),stream);

	W_final = gpuArrayAllocRM(DataType::FLOAT,DDim(NUM_CLASSES*NUM_CLASSIFIERS,BOTTLENECK),stream);
	Ptr<DistArrayHandle> W_final_load = loadDistArray(model+"_W_final",DDim(1,1,1,1),stream);
	copy(W_final_load->localHandle(),W_final,stream);
	B_final = gpuArrayAllocRM(DataType::FLOAT,DDim(1,W_final->dim(0)),stream);
	Ptr<DistArrayHandle> B_final_load = loadDistArray(model+"_B_final",DDim(1,1,1,1),stream);
	copy(B_final_load->localHandle(),B_final,stream);
	mult_linear = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,W_final->dim(0)),stream);

	reduce_col = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,1,NUM_CLASSIFIERS),stream);
	indexArray = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,NUM_CLASSES,NUM_CLASSIFIERS),stream);
	for(int i=0;i<NUM_CLASSES;i++){
		gpuSet(i,indexArray->view(DDim(0,i,0),DDim(BATCH_SIZE,1,NUM_CLASSIFIERS)),stream);
	}
	tmpArray = gpuArrayAllocRM(DataType::FLOAT,indexArray->dim(),stream);
	labels = gpuArrayAllocRM(DataType::FLOAT,reduce_col->dim(),stream);

	allocScratchArray(data_buf,mean_lcn,filt_LCN,MEAN_STEP2,data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
	allocScratchArray(mean_lcn_sqr,divide_lcn,filt_LCN,MEAN_STEP2,mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
	allocScratchArray(divide_lcn,filt_1,W_1,FILT1_STEPY,divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	allocScratchArray(pool_1,filt_2,W_2,FILT2_STEPY,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	allocScratchArray(pool_2,feat,W_3,FILT3_STEPY,pool_2_scratch,feat_scratch,stream,W3_SPLITS);
    
    host_output = hostArrayAllocRM(DataType::FLOAT,DDim(1,NUM_CLASSIFIERS),stream);

	synchronizeStream(stream);
}
LanePredictor_honda::~LanePredictor_honda(){
	fastCppShutdown();
}

Ptr<ArrayViewHandle> LanePredictor_honda::processImage(const Ptr<ArrayViewHandle>& img){
    copy(img,data_buf,stream);
	gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,data_buf,mean_lcn,
		data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
	gpuMinus(data_buf->view(MEAN_POS,mean_lcn->dim()),mean_lcn,mean_lcn,stream);

	gpuSquare(mean_lcn,mean_lcn_sqr,stream);
	gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,mean_lcn_sqr,divide_lcn,
		mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
	gpuPlusScalar(divide_lcn,DIVIDE_EPS,divide_lcn,stream);
	gpuSqrt(divide_lcn,divide_lcn,stream);
	gpuDivide(mean_lcn->view(DIVIDE_POS,divide_lcn->dim()),divide_lcn,divide_lcn,stream);

	gpuFilterTimesLarge(W_1,false,FILT1_STEPZ,FILT1_STEPX,FILT1_STEPY,divide_lcn,filt_1,
		divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	gpuBinaryOp(filt_1,filt_1,nonlin_1,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_1,2,3,SUBSAMP1_DIMX,SUBSAMP1_DIMY,SUBSAMP1_STEPX,SUBSAMP1_STEPY,pool_1,stream);
	gpuTimesScalar(pool_1,1.0/(SUBSAMP1_DIMX*SUBSAMP1_DIMY),pool_1,stream);

	gpuFilterTimesLarge(W_2,false,FILT2_STEPZ,FILT2_STEPX,FILT2_STEPY,pool_1,filt_2,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	gpuBinaryOp(filt_2,filt_2,nonlin_2,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_2,2,3,SUBSAMP2_DIMX,SUBSAMP2_DIMY,SUBSAMP2_STEPX,SUBSAMP2_STEPY,pool_2,stream);
	gpuTimesScalar(pool_2,1.0/(SUBSAMP2_DIMX*SUBSAMP2_DIMY),pool_2,stream);

	gpuFilterTimesLarge(W_3,false,FILT3_STEPZ,FILT3_STEPX,FILT3_STEPY,pool_2,feat,pool_2_scratch,feat_scratch,stream,W3_SPLITS);

	gpuGEMM('n','t',1.0f,reshape(feat,DDim(BATCH_SIZE,feat->dim(1)*feat->dim(2)*feat->dim(3)),stream),
		reshape(W_bn,DDim(BOTTLENECK,W_bn->dim(1)*W_bn->dim(2)*W_bn->dim(3)),stream),0.0f,feat_bn,stream);
	gpuPlus(feat_bn,B_bn,feat_bn,stream);

	gpuGEMM('n','t',1.0f,feat_bn,W_final,0.0f,mult_linear,stream);
	gpuPlus(mult_linear,B_final,mult_linear,stream);
	Ptr<ArrayViewHandle> prob = reshape(mult_linear,DDim(BATCH_SIZE,NUM_CLASSES,NUM_CLASSIFIERS),stream);

	gpuMax(prob,1,reduce_col,stream);
	gpuMinus(prob,reduce_col,prob,stream);
	gpuExp(prob,prob,stream);
	gpuUnaryOp(prob,MIN_ACTIVATION,prob,"fmaxf(a,b)",stream);
	gpuSum(prob,1,reduce_col,stream);
	gpuDivide(prob,reduce_col,prob,stream);
	gpuUnaryOp(prob,MIN_ACTIVATION,prob,"fmaxf(a,b)",stream);
	gpuUnaryOp(prob,MAX_ACTIVATION,prob,"fminf(a,b)",stream);

	//find pixel output
	gpuMax(prob,1,reduce_col,stream);
	gpuEquals(prob,reduce_col,tmpArray,stream);
	gpuTimes(indexArray,tmpArray,tmpArray,stream);
	gpuMax(tmpArray,1,labels,stream);
	gpuPlusScalar(labels,-SUBPIXEL_WINDOW/2,reduce_col,stream);
	gpuGreaterEqual(indexArray,reduce_col,tmpArray,stream);
	gpuTimes(tmpArray,prob,prob,stream);
	gpuPlusScalar(labels,SUBPIXEL_WINDOW/2,reduce_col,stream);
	gpuLessEqual(indexArray,reduce_col,tmpArray,stream);
	gpuTimes(prob,tmpArray,prob,stream);
	gpuSum(prob,1,reduce_col,stream);
	gpuTimes(prob,indexArray,prob,stream);
	gpuSum(prob,1,labels,stream);
	gpuDivide(labels,reduce_col,labels,stream); 

	Ptr<ArrayViewHandle> label_output = reshape(labels,DDim(1,NUM_CLASSIFIERS),stream);
    copy(label_output,host_output,stream);

	return host_output;
}
