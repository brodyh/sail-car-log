#include "hip/hip_runtime.h"
#include "FastCpp.h"
#include "gpu/kernels/GpuKernels.h"
#include <fstream>
#include <iostream>
#include "gpu/GpuBlockOps.h"
#include "gpu/GpuMath.h"
#include "io/ArrayIO.h"
#include "hip/hip_runtime_api.h"
#include "LanePredictor.h"

LanePredictor::LanePredictor(int* argc, char* argv[],int stream_num){
	stream = stream_num;
	model = argv[1];
	fastCppInit(argc,&argv,stream);

	data_buf = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,IMG_DIMZ,IMG_DIMX,IMG_DIMY), stream);
	mean_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,MEAN_OUTPUTZ,MEAN_OUTPUTX,MEAN_OUTPUTY),stream);
	mean_lcn_sqr = gpuArrayAllocRM(DataType::FLOAT, mean_lcn->dim(),stream);
	divide_lcn = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,DIVIDE_OUTPUTZ,DIVIDE_OUTPUTX,DIVIDE_OUTPUTY),stream);
	filt_LCN = gpuArrayAllocRM(DataType::FLOAT,DDim(IMG_DIMZ,1,1,IMG_DIMZ,MEAN_SIZE1,MEAN_SIZE2),stream);
	init_filt_LCN();

	W_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_1,1,1,FILT1_DIMZ,FILT1_DIMX,FILT1_DIMY), stream);
	Ptr<DistArrayHandle> W_1_load = loadDistArray(model+"_W_1",DDim(1,1,1,1,1,1),stream);
	copy(W_1_load->localHandle(),W_1,stream);
	filt_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT1_OUTPUTZ,FILT1_OUTPUTX,FILT1_OUTPUTY), stream);
	nonlin_1 = gpuArrayAllocRM(DataType::FLOAT, filt_1->dim(),stream);
	pool_1 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP1_OUTPUTZ,SUBSAMP1_OUTPUTX,SUBSAMP1_OUTPUTY),stream);

	W_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_2,1,1,FILT2_DIMZ,FILT2_DIMX,FILT2_DIMY),stream);
	Ptr<DistArrayHandle> W_2_load = loadDistArray(model+"_W_2",DDim(1,1,1,1,1,1),stream);
	copy(W_2_load->localHandle(),W_2,stream);
	filt_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT2_OUTPUTZ,FILT2_OUTPUTX,FILT2_OUTPUTY),stream);
	nonlin_2 = gpuArrayAllocRM(DataType::FLOAT, filt_2->dim(),stream);
	pool_2 = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,SUBSAMP2_OUTPUTZ,SUBSAMP2_OUTPUTX,SUBSAMP2_OUTPUTY),stream);

	W_3 = gpuArrayAllocRM(DataType::FLOAT, DDim(MAPS_3,1,1,FILT3_DIMZ,FILT3_DIMX,FILT3_DIMY),stream);
	Ptr<DistArrayHandle> W_3_load = loadDistArray(model+"_W_3",DDim(1,1,1,1,1,1),stream);
	copy(W_3_load->localHandle(),W_3,stream);
	feat = gpuArrayAllocRM(DataType::FLOAT, DDim(BATCH_SIZE,FILT3_OUTPUTZ,FILT3_OUTPUTX,FILT3_OUTPUTY),stream);

	W_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BOTTLENECK,feat->dim(1),feat->dim(2),feat->dim(3)),stream);
	Ptr<DistArrayHandle> W_bn_load = loadDistArray(model+"_W_bn",DDim(1,1,1,1),stream);
	copy(W_bn_load->localHandle(),W_bn,stream);
	B_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(1,BOTTLENECK),stream);
	Ptr<DistArrayHandle> B_bn_load = loadDistArray(model+"_B_bn",DDim(1,1,1,1),stream);
	copy(B_bn_load->localHandle(),B_bn,stream);
	feat_bn = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,BOTTLENECK),stream);

	W_final = gpuArrayAllocRM(DataType::FLOAT,DDim(NUM_CLASSES*NUM_CLASSIFIERS,BOTTLENECK),stream);
	Ptr<DistArrayHandle> W_final_load = loadDistArray(model+"_W_final",DDim(1,1,1,1),stream);
	copy(W_final_load->localHandle(),W_final,stream);
	B_final = gpuArrayAllocRM(DataType::FLOAT,DDim(1,W_final->dim(0)),stream);
	Ptr<DistArrayHandle> B_final_load = loadDistArray(model+"_B_final",DDim(1,1,1,1),stream);
	copy(B_final_load->localHandle(),B_final,stream);
	mult_linear = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,W_final->dim(0)),stream);

	reduce_col = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,1,NUM_CLASSIFIERS),stream);
	indexArray = gpuArrayAllocRM(DataType::FLOAT,DDim(BATCH_SIZE,NUM_CLASSES,NUM_CLASSIFIERS),stream);
	for(int i=0;i<NUM_CLASSES;i++){
		gpuSet(i,indexArray->view(DDim(0,i,0),DDim(BATCH_SIZE,1,NUM_CLASSIFIERS)),stream);
	}
	tmpArray = gpuArrayAllocRM(DataType::FLOAT,indexArray->dim(),stream);
	labels = gpuArrayAllocRM(DataType::FLOAT,reduce_col->dim(),stream);

	allocScratchArray(data_buf,mean_lcn,filt_LCN,MEAN_STEP1,data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
	allocScratchArray(mean_lcn_sqr,divide_lcn,filt_LCN,MEAN_STEP1,mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
	allocScratchArray(divide_lcn,filt_1,W_1,FILT1_STEPX,divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	allocScratchArray(pool_1,filt_2,W_2,FILT2_STEPX,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	allocScratchArray(pool_2,feat,W_3,FILT3_STEPX,pool_2_scratch,feat_scratch,stream,W3_SPLITS);
    
    host_output = hostArrayAllocRM(DataType::FLOAT,DDim(1,NUM_CLASSIFIERS),stream);

	synchronizeStream(stream);
}
LanePredictor::~LanePredictor(){
	fastCppShutdown();
}

Ptr<ArrayViewHandle> LanePredictor::processImage(const Ptr<ArrayViewHandle>& img){
    copy(img,data_buf,stream);
	gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,data_buf,mean_lcn,
		data_buf_scratch,mean_lcn_scratch,stream,MEAN_LCN_SPLITS);
	gpuMinus(data_buf->view(MEAN_POS,mean_lcn->dim()),mean_lcn,mean_lcn,stream);

	gpuSquare(mean_lcn,mean_lcn_sqr,stream);
	gpuFilterTimesLarge(filt_LCN,false,IMG_DIMZ,MEAN_STEP1,MEAN_STEP2,mean_lcn_sqr,divide_lcn,
		mean_lcn_sqr_scratch,divide_lcn_scratch,stream,DIVIDE_LCN_SPLITS);
	gpuPlusScalar(divide_lcn,DIVIDE_EPS,divide_lcn,stream);
	gpuSqrt(divide_lcn,divide_lcn,stream);
	gpuDivide(mean_lcn->view(DIVIDE_POS,divide_lcn->dim()),divide_lcn,divide_lcn,stream);

	gpuFilterTimesLarge(W_1,false,FILT1_STEPZ,FILT1_STEPX,FILT1_STEPY,divide_lcn,filt_1,
		divide_lcn_in_scratch,filt_1_scratch,stream,W1_SPLITS);
	gpuBinaryOp(filt_1,filt_1,nonlin_1,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_1,2,3,SUBSAMP1_DIMX,SUBSAMP1_DIMY,SUBSAMP1_STEPX,SUBSAMP1_STEPY,pool_1,stream);
	gpuTimesScalar(pool_1,1.0/(SUBSAMP1_DIMX*SUBSAMP1_DIMY),pool_1,stream);

	gpuFilterTimesLarge(W_2,false,FILT2_STEPZ,FILT2_STEPX,FILT2_STEPY,pool_1,filt_2,pool_1_scratch,filt_2_scratch,stream,W2_SPLITS);
	gpuBinaryOp(filt_2,filt_2,nonlin_2,THRESHOLD_NL,stream);
	gpuLocalSum2D(nonlin_2,2,3,SUBSAMP2_DIMX,SUBSAMP2_DIMY,SUBSAMP2_STEPX,SUBSAMP2_STEPY,pool_2,stream);
	gpuTimesScalar(pool_2,1.0/(SUBSAMP2_DIMX*SUBSAMP2_DIMY),pool_2,stream);

	gpuFilterTimesLarge(W_3,false,FILT3_STEPZ,FILT3_STEPX,FILT3_STEPY,pool_2,feat,pool_2_scratch,feat_scratch,stream,W3_SPLITS);

	gpuGEMM('n','t',1.0f,reshape(feat,DDim(BATCH_SIZE,feat->dim(1)*feat->dim(2)*feat->dim(3)),stream),
		reshape(W_bn,DDim(BOTTLENECK,W_bn->dim(1)*W_bn->dim(2)*W_bn->dim(3)),stream),0.0f,feat_bn,stream);
	gpuPlus(feat_bn,B_bn,feat_bn,stream);

	gpuGEMM('n','t',1.0f,feat_bn,W_final,0.0f,mult_linear,stream);
	gpuPlus(mult_linear,B_final,mult_linear,stream);
	Ptr<ArrayViewHandle> prob = reshape(mult_linear,DDim(BATCH_SIZE,NUM_CLASSES,NUM_CLASSIFIERS),stream);

	gpuMax(prob,1,reduce_col,stream);
	gpuMinus(prob,reduce_col,prob,stream);
	gpuExp(prob,prob,stream);
	gpuUnaryOp(prob,MIN_ACTIVATION,prob,"fmaxf(a,b)",stream);
	gpuSum(prob,1,reduce_col,stream);
	gpuDivide(prob,reduce_col,prob,stream);
	gpuUnaryOp(prob,MIN_ACTIVATION,prob,"fmaxf(a,b)",stream);
	gpuUnaryOp(prob,MAX_ACTIVATION,prob,"fminf(a,b)",stream);

	//find pixel output
	gpuMax(prob,1,reduce_col,stream);
	gpuEquals(prob,reduce_col,tmpArray,stream);
	gpuTimes(indexArray,tmpArray,tmpArray,stream);
	gpuMax(tmpArray,1,labels,stream);
	gpuPlusScalar(labels,-SUBPIXEL_WINDOW/2,reduce_col,stream);
	gpuGreaterEqual(indexArray,reduce_col,tmpArray,stream);
	gpuTimes(tmpArray,prob,prob,stream);
	gpuPlusScalar(labels,SUBPIXEL_WINDOW/2,reduce_col,stream);
	gpuLessEqual(indexArray,reduce_col,tmpArray,stream);
	gpuTimes(prob,tmpArray,prob,stream);
	gpuSum(prob,1,reduce_col,stream);
	gpuTimes(prob,indexArray,prob,stream);
	gpuSum(prob,1,labels,stream);
	gpuDivide(labels,reduce_col,labels,stream); 

	Ptr<ArrayViewHandle> label_output = reshape(labels,DDim(1,NUM_CLASSIFIERS),stream);
    copy(label_output,host_output,stream);

	return host_output;
}
void LanePredictor::init_filt_LCN(){
	Ptr<ArrayViewHandle> filt_vals = hostArrayAllocRM(DataType::FLOAT,DDim(MEAN_SIZE1,MEAN_SIZE2),stream);
	double fsum = 0.0;
	int hp = (MEAN_SIZE1-1)/2;
	for(int i=0;i<MEAN_SIZE1;i++){
		for(int j=0;j<MEAN_SIZE2;j++){
			int dx = i-hp;
			int dy = j-hp;
			double c = expf(-0.5*(dx*dx+dy*dy)/(MEAN_SIGMA*MEAN_SIGMA));
			cpuSet(c,filt_vals->view(DDim(i,j),DDim(1,1)),stream);
			fsum += c;
		}
	}
	cpuTimesScalar(filt_vals,1.0/fsum/IMG_DIMZ,filt_vals,stream);
	for(int i=0;i<IMG_DIMZ;i++){
		for(int j=0;j<IMG_DIMZ;j++){
			copy(filt_vals,filt_LCN->view(DDim(i,0,0,j,0,0),DDim(1,1,1,1,MEAN_SIZE1,MEAN_SIZE2)),stream);
		}
	}
}

void LanePredictor::allocScratchArray(  const Ptr<ArrayViewHandle>& input,
                                        const Ptr<ArrayViewHandle>& output,
                                        const Ptr<ArrayViewHandle>& filters,
                                        int stepx,
                                        Ptr<ArrayViewHandle>& inputScratch,
                                        Ptr<ArrayViewHandle>& outputScratch,
                                        int stream,
                                        int num_splits){
	if(num_splits==0){
		num_splits = clp2((output->dim(2)*output->dim(3)-1)/65535+1);
	}
	int split_width = output->dim(2)/num_splits;
	if(split_width*num_splits != output->dim(2)){
		split_width += 1;
	}
	inputScratch =  gpuArrayAllocRM(input->dataType(),DDim(num_splits,input->dim(1),filters->dim(4)+(split_width-1)*stepx,
		input->dim(3)),stream);
	outputScratch = gpuArrayAllocRM(output->dataType(),DDim(num_splits,output->dim(1),split_width,output->dim(3)),stream);
}

void LanePredictor::gpuFilterTimesLarge(  const Ptr<ArrayViewHandle>& filters,
                                          bool transpose,
                                          int stepz, int stepx, int stepy,
                                          const Ptr<ArrayViewHandle>& input,
                                          const Ptr<ArrayViewHandle>& output,
                                          const Ptr<ArrayViewHandle>& inputScratch,
                                          const Ptr<ArrayViewHandle>& outputScratch,
                                          int stream,
                                          int num_splits){
	if(num_splits==0){
		num_splits = clp2((output->dim(2)*output->dim(3)-1)/65535+1);
	}
	int out_split_width = output->dim(2)/num_splits;
	if(out_split_width*num_splits != output->dim(2)){
		out_split_width += 1;
	}
	int in_split_width = filters->dim(4)+(out_split_width-1)*stepx;
	Ptr<ArrayViewHandle> input_split;
	for(int i=0;i<num_splits;i++){
		int in_split_offset = i*out_split_width*stepx;
	  input_split = input->view(DDim(0,0,in_split_offset,0),DDim(input->dim(0),input->dim(1),in_split_width,input->dim(3)));
		gpuCopy(input_split,inputScratch->view(DDim(i,0,0,0),input_split->dim()),stream);
		if((in_split_width+in_split_offset) > input->dim(2)){
			break;
		}
	}
	gpuFilterTimes(filters,transpose,stepz,stepx,stepy,inputScratch,outputScratch,stream);
	Ptr<ArrayViewHandle> output_split;
	for(int i=0;i<num_splits;i++){
		if(out_split_width*(i+1) > output->dim(2)){
			output_split = output->view(DDim(0,0,i*out_split_width,0),DDim(output->dim(0),output->dim(1),
				output->dim(2)-i*out_split_width,output->dim(3)));
			gpuCopy(outputScratch->view(DDim(i,0,0,0),output_split->dim()),output_split,stream);
			break;
		}else{
			output_split = output->view(DDim(0,0,i*out_split_width,0),DDim(output->dim(0),output->dim(1),
				out_split_width,output->dim(3)));
			gpuCopy(outputScratch->view(DDim(i,0,0,0),output_split->dim()),output_split,stream);
		}
	}
}


